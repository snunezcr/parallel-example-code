#include "hip/hip_runtime.h"
/*
 * Instituto Tecnologico de Costa Rica
 * Centro de Investigaciones en Computacion
 * 
 * Asesoria-Practica en Computacion Paralela
 * Instituto Costarricense de Electricidad
 * Julio-Agosto 2011
 * 
 * Autor: Santiago Nunez Corrales
 * Programa: sort bitonico en CUDA C
 */


#include <math.h>

#define DEF_TAM 64

__global__ void comparar( int* b, int* a, int largo, int offset, int N) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int mi_valor;

	a[idx] = b[idx];

	if (((idx/largo)%2) == 0)
		if (((idx/offset)%2) == 0)
			if (a[idx] < a[idx + offset])
				mi_valor = a[idx];
			else
				mi_valor = a[idx + offset];
		else
			if (a[idx] < a[idx - offset])
				mi_valor = a[idx - offset];
			else
				mi_valor = a[idx];
	else
		if (((idx/offset)%2) == 0)
			if (a[idx] < a[idx + offset])
				mi_valor = a[idx + offset];
			else
				mi_valor = a[idx];
		else
			if (a[idx] < a[idx - offset])
				mi_valor = a[idx];
			else
				mi_valor = a[idx - offset];

	b[idx] = mi_valor;
}

int main(int argc, char *argv[]) {
	int *a;
	int *b;
	int *c;
	int i;
	int largo;
	int offset

	hipMalloc((void**)&a, sizeof(int)*DEF_TAM);
	hipMalloc((void**)&b, sizeof(int)*DEF_TAM);

	c = (int*) malloc(sizeof(int) * 64);

	for (i = 0; i < DEF_TAM; i++)
		c[i] = random() % 1024;

	for (i = 0; i < DEF_TAM; i++)
		printf(" %d", c[i]);
	printf("\n");

	hipMemcpy(b, c, sizeof(int)*DEF_TAM, hipMemcpyHostToDevice);

	for (largo = 2; largo <= DEF_TAM; largo+= largo)
		for (offset = largo/2; offset > 0; offset /= 2)
			comparar<<<1, DEF_TAM>>>(b, a, largo, offset, DEF_TAM);


	hipMemcpy(c, b, sizeof(int)*DEF_TAM, hipMemcpyDeviceToHost));

	for (int i = 0; i < DEF_TAM; ++i)
		printf(" %d ", c[i]);
	printf("\n");

	hipFree(a);
	hipFree(b);
	free(c);

	return 0;
}

