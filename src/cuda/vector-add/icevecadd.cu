/*
 * Instituto Tecnologico de Costa Rica
 * Centro de Investigaciones en Computacion
 * 
 * Asesoria-Practica en Computacion Paralela
 * Instituto Costarricense de Electricidad
 * Julio-Agosto 2011
 * 
 * Autor: Santiago Nunez Corrales
 * Programa: suma de vectores en CUDA C
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define DEF_TAM 10

__global__ void suma_vec(float *a, float *b, float *c) {
	/* threadIdx.x es el identificador del hilo en la dimension x */
	int idx = threadIdx.x;
	a[idx] = 0;
	b[idx] = idx;
	c[idx] = a[idx] + b[idx];
}

int main (int argc, char *argv[]) {
	int n;
	int i;
	float a[DEF_TAM];
	float b[DEF_TAM];
	float c[DEF_TAM];
	float *ptr_dev_a;
	float *ptr_dev_b;
	float *ptr_dev_c;
	int mem_reservada;

	n = DEF_TAM;
	mem_reservada = DEF_TAM * sizeof(float);

	hipMalloc((void **)&ptr_dev_a, mem_reservada);
	hipMalloc((void **)&ptr_dev_b, mem_reservada);
	hipMalloc((void **)&ptr_dev_c, mem_reservada);

	hipMemcpy(ptr_dev_a, a, mem_reservada, hipMemcpyHostToDevice);
	hipMemcpy(ptr_dev_b, b, mem_reservada, hipMemcpyHostToDevice);

	suma_vec<<<1, n>>>(ptr_dev_a, ptr_dev_b, ptr_dev_c);

	hipMemcpy(c, ptr_dev_c, mem_reservada, hipMemcpyDeviceToHost);

	for (i = 0; i < DEF_TAM; i++)
		printf("c[%d] = %f\n", i, c[i]);

	hipFree(ptr_dev_a);
	hipFree(ptr_dev_b);
	hipFree(ptr_dev_c);

	return 0;
}


